#include <stdio.h>
#include <stdlib.h>
#include <string.h>
//#include <omp.h>
#include <hip/hip_runtime.h>


const char ARQUIVO_DNA_1[] = {"src/DNA1.txt"};
const char ARQUIVO_DNA_2[] = {"src/DNA2.txt"};

char *DNA, *combinacoesDNA;

int *inicioProcesso, *countCombinacoesDNA;

char combinacoesDNAtxt[24][5] = {"ACGT", "ACTG", "AGCT" , "AGTC", 
                                "ATCG", "ATGC", "CAGT", "CATG", 
                                "CGAT", "CGTA", "CTAG", "CTGA", 
                                "GACT", "GATC", "GCAT", "GCTA", 
                                "GTAC", "GTCA", "TACG", "TAGC", 
                                "TCAG", "TCGA", "TGAC", "TGCA"};
                            
void lerArquivo(const char *nomeArquivo, const int tamanho);
int tamanhoArquivo(const char *nomeArquivo);

__device__ int my_strcmp(const char *str_a, const char *str_b, unsigned len = 5)
{
    int match = 0;
    unsigned i = 0;
    unsigned done = 0;
    while ((i < len) && (match == 0) && !done)
    {
      if ((str_a[i] == 0) || (str_b[i] == 0))
      {
        done = 1;
      } 
      else if (str_a[i] != str_b[i])
      {
        match = i+1;
        if (((int)str_a[i] - (int)str_b[i]) < 0)
        {
        match = 0 - (i + 1);
        }
      }
      i++;
    }
    return match;
}

__global__ void pesquisar(char *dna, char *combinacoes, int *inicio, int *countCombinacoes, int tamanhoDNA, unsigned totalProcessos)
{
    int idProcesso = blockIdx.x * blockDim.x + threadIdx.x;
    if(idProcesso < totalProcessos)
    {
        int i;
    
        char busca[5];
        char combinacao[5];

        int count;
        for(count = inicio[idProcesso]; count < inicio[idProcesso + 1]; count++)
        {
            
            busca[0] = dna[count];
            busca[1] = dna[count + 1];
            busca[2] = dna[count + 2];
            busca[3] = dna[count + 3];
            
            for(i = 0; i < 96; i+=4)
            {
                combinacao[0] = combinacoes[i];
                combinacao[1] = combinacoes[i + 1];
                combinacao[2] = combinacoes[i + 2];
                combinacao[3] = combinacoes[i + 3];
                
                if(my_strcmp(busca, combinacao) == 0)
                {
                    // Add no vetor
                    atomicAdd(&countCombinacoes[i/4],1);
                    break;
                }  
            }
        }
    }
}

__global__ void compararEncontrados(char *dna, char *combinacoes, int *inicio, int *maisEncontrados, int *countmaisEncontrados, int tamanhoDNA, int totalProcessos)
{
    int idProcesso = blockIdx.x * blockDim.x + threadIdx.x;
    if(idProcesso < totalProcessos)
    {

        int i;
    
        char busca[5];
        char combinacao[5];
        
        int count;
        for(count = inicio[idProcesso]; count < inicio[idProcesso + 1]; count++)
        {
            
            busca[0] = dna[count];
            busca[1] = dna[count + 1];
            busca[2] = dna[count + 2];
            busca[3] = dna[count + 3];
            
            for(i = 0; i < 17; i+=4)
            {   
                int tempIndex = maisEncontrados[i/4] * 4;
                combinacao[0] = combinacoes[tempIndex];
                combinacao[1] = combinacoes[tempIndex + 1];
                combinacao[2] = combinacoes[tempIndex + 2];
                combinacao[3] = combinacoes[tempIndex + 3];
                
                if(my_strcmp(busca, combinacao) == 0)
                {
                    // Add no vetor
                    atomicAdd(&countmaisEncontrados[i/4],1);  
                    break;
                }
            }
        }
    }
}


int main(int argc, char *argv[0])
{
    if((argv[1] == NULL) || (argv[2] == NULL))
    {
        printf("\n\nInsira o número de blocos e threads! \n\n");
        return(1);
    }

    int *countMaisEncontrados;

    int numBlocos = atoi(argv[1]);
    int numThreads = atoi(argv[2]);
    int totalProcessos = numBlocos * numThreads;

    const int tamanhoDNA = tamanhoArquivo(ARQUIVO_DNA_1);

    hipMallocManaged(&DNA, tamanhoDNA * sizeof(char));
    hipMallocManaged(&combinacoesDNA, 96 * sizeof(char));
    hipMallocManaged(&inicioProcesso, (totalProcessos + 1) * sizeof(int));
    hipMallocManaged(&countCombinacoesDNA, 24 * sizeof(int));
    hipMallocManaged(&countMaisEncontrados, 5 * sizeof(int));
    
    strcpy(combinacoesDNA,"ACGTACTGAGCTAGTCATCGATGCCAGTCATGCGATCGTACTAGCTGAGACTGATCGCATGCTAGTACGTCATACGTAGCTCAGTCGATGACTGCA");
    
    lerArquivo(ARQUIVO_DNA_1, tamanhoDNA);

    // Definindo a posição inicial de cada processo
    int i;
    inicioProcesso[0] = 0;
    for(i = 0; i < totalProcessos; i++)
    {
        inicioProcesso[i + 1] = inicioProcesso[i] + tamanhoDNA / totalProcessos;
    }

    /*
    <<<numDeBlocos, numDeThreadsPorBloco>>>
    gtx 960 16 blocos 64 threads
    gtx 1050 12 blocos 64 threads
    */

    pesquisar <<<numBlocos, numThreads>>> (DNA, combinacoesDNA, inicioProcesso, countCombinacoesDNA, tamanhoDNA, totalProcessos);
    hipDeviceSynchronize();
    
    int *maisEncontrados;
    hipMallocManaged(&maisEncontrados, 5 * sizeof(int));
    maisEncontrados[0] = 0;
    // Encontra as 5 combinações mais achadas pegando os seus indices
    for (int count = 0; count < 5; count++)
    {
        if (count == 0)
        {
            for (i = 0; i < 24; i++)
            {
                if (countCombinacoesDNA[maisEncontrados[count]] < countCombinacoesDNA[i])
                {
                    maisEncontrados[count] = i;
                }
            }
        }
        else
        {
            for (i = 0; i < 24; i++)
            {
                if (countCombinacoesDNA[maisEncontrados[count]] < countCombinacoesDNA[i] && countCombinacoesDNA[maisEncontrados[count - 1]] > countCombinacoesDNA[i])
                {
                    maisEncontrados[count] = i;
                }
            }
        }
    }

    lerArquivo(ARQUIVO_DNA_2, tamanhoDNA);

    compararEncontrados<<<numBlocos, numThreads>>>(DNA, combinacoesDNA, inicioProcesso, maisEncontrados, countMaisEncontrados, tamanhoDNA, totalProcessos);
    hipDeviceSynchronize();

    hipFree(DNA);
    hipFree(countCombinacoesDNA);
    hipFree(combinacoesDNA);
    hipFree(inicioProcesso);
    hipFree(maisEncontrados);
    hipFree(countMaisEncontrados);

    return 0;
}

void lerArquivo(const char *nomeArquivo, const int tamanho)
{   
    /*
    omp_set_num_threads(12);
    #pragma omp parallel
    {
        FILE *file;
        file = fopen(nomeArquivo, "r");
    
        int i;
        char busca[2];
        // Leitura do arquivo DNA1
        #pragma omp for private (i) 
        fseek(file, i, SEEK_SET);
        for (i = 0; i < tamanho; i++)
        {
            //fseek(file, i, SEEK_SET);
            fgets(busca, 2, file);
            DNA[i] = busca[0];
            //fscanf(file, "%1c", &DNA[i]);
        }
        fclose(file);
    }
    */
    
    
    FILE *file;
    file = fopen(nomeArquivo,"r");
    
    int i;
    // Leitura do arquivo DNA1
    for(i = 0; i < tamanho; i++)
    {
        fscanf(file, "%1c", &DNA[i]);
    }
    fclose(file);
    
    
}

int tamanhoArquivo(const char *nomeArquivo)
{
    FILE *file;
    file = fopen(nomeArquivo, "r");
    fseek(file, 0, SEEK_END);
    int size;
    size = ftell(file);
    fclose(file);
    return size;
}

